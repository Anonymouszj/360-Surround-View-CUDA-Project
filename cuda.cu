#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <opencv2/opencv.hpp>
using namespace std;
using namespace cv;

struct float10 {
  float x[10];
};

struct ptr4 {
  uchar3* v[4];
};

static __device__ __forceinline__ uchar3 belend(uchar3 a, uchar3 b, float w) {
  return make_uchar3(a.x * w + b.x * (1 - w), a.y * w + b.y * (1 - w),
                     a.z * w + b.z * (1 - w));
}

static __global__ void surround_kernel(const float10* table, int w, int h,
                                       ptr4 images, int iw, int ih,
                                       uchar3* output) {
  int ix = blockDim.x * blockIdx.x + threadIdx.x;
  int iy = blockDim.y * blockIdx.y + threadIdx.y;
  if (ix >= w || iy >= h) return;

  int pos = iy * w + ix;
  float10 item = table[pos];
  int flag = item.x[0];
  float weight = item.x[1];

  if (flag == -1) return;
  if (flag < 4) {
    int x = item.x[2 + flag * 2 + 0];
    int y = item.x[2 + flag * 2 + 1];

    output[pos] = images.v[flag][y * iw + x];
  } else {
    const int idxs[][2] = {{2, 1}, {0, 3}, {0, 1}, {2, 3}};
    int a = idxs[flag - 4][0];
    int b = idxs[flag - 4][1];
    int ax = item.x[2 + a * 2 + 0];
    int ay = item.x[2 + a * 2 + 1];
    int bx = item.x[2 + b * 2 + 0];
    int by = item.x[2 + b * 2 + 1];
    output[pos] =
        belend(images.v[a][ay * iw + ax], images.v[b][by * iw + bx], weight);
  }
}

class Surrounder {
 public:
  virtual ~Surrounder() { destroy(); }

  bool load(const std::string& file, int w, int h, int numcam, int camw,
            int camh) {
    FILE* f = fopen(file.c_str(), "rb");
    if (f == nullptr) {
      printf("Failed to load table: %s\n", file.c_str());
      return false;
    }

    fseek(f, 0, SEEK_END);
    size_t size = ftell(f);
    fseek(f, 0, SEEK_SET);

    if (size != w * h * 10 * sizeof(float)) {
      printf("Invalid table file.\n");
      fclose(f);
      return false;
    }

    unsigned char* table_host = new unsigned char[size];
    fread(table_host, 1, size, f);
    fclose(f);

    w_ = w;
    h_ = h;
    camw_ = camw;
    camh_ = camh;
    output_.create(h_, w_, CV_8UC3);

    for (int i = 0; i < numcam; ++i) {
      unsigned char* device_ptr = nullptr;
      hipMalloc(&device_ptr, camw * camh * 3 * sizeof(unsigned char));
      images_device_.push_back(device_ptr);
    }

    hipMalloc(&output_view_, w_ * h_ * 3 * sizeof(unsigned char));
    hipMalloc(&table_, size);
    hipMemcpy(table_, table_host, size, hipMemcpyHostToDevice);
    delete[] table_host;
    return true;
  }

  cv::Mat forward(const std::vector<cv::Mat>& images,
                  hipStream_t stream = nullptr) {
    if (images.size() != images_device_.size()) {
      printf("Mismatched image size.\n");
      return cv::Mat();
    }

    for (int i = 0; i < images.size(); ++i) {
      auto& image = images[i];
      if (image.cols != camw_ || image.rows != camh_) {
        printf("Invalid image size: %d x %d\n", image.cols, image.rows);
        return cv::Mat();
      }

      hipMemcpyAsync(images_device_[i], image.data,
                      image.cols * image.rows * 3 * sizeof(unsigned char),
                      hipMemcpyHostToDevice, stream);
    }

    if (images.size() != 4) {
      printf("Unsupported image size.\n");
      return cv::Mat();
    }

    ptr4 images_ptr;
    memcpy(images_ptr.v, images_device_.data(), sizeof(images_device_[0]) * 4);
    dim3 block(32, 32);
    dim3 grid((w_ + block.x - 1) / block.x, (h_ + block.y - 1) / block.y);
    surround_kernel<<<grid, block, 0, stream>>>(
        table_, w_, h_, images_ptr, camw_, camh_, (uchar3*)output_view_);

    hipMemcpyAsync(output_.data, output_view_,
                    output_.rows * output_.cols * 3 * sizeof(unsigned char),
                    hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);
    return output_;
  }

 private:
  void destroy() {
    for (int i = 0; i < images_device_.size(); ++i) {
      hipFree(images_device_[i]);
    }
    images_device_.clear();

    if (table_) {
      hipFree(table_);
      table_ = nullptr;
    }

    if (output_view_) {
      hipFree(output_view_);
      output_view_ = nullptr;
    }
  }

 private:
  std::vector<unsigned char*> images_device_;
  float10* table_ = nullptr;
  unsigned char* output_view_ = nullptr;
  cv::Mat output_;
  int w_ = 0;
  int h_ = 0;
  int camw_ = 0;
  int camh_ = 0;
};

int main() {
  Surrounder surround;
  if (!surround.load("surround_view.binary", 1200, 1600, 4, 960, 640)) {
    return -1;
  }

  const char* image_names[] = {"front", "left", "back", "right"};
  std::vector<cv::Mat> images;

  for (int i = 0; i < 4; ++i) {
    images.emplace_back(
        cv::imread(cv::format("images/%s.png", image_names[i])));
  }

  auto output = surround.forward(images);
  cv::imwrite("surround.jpg", output);
  printf("hello %d x %d\n", images[0].cols, images[0].rows);
  return 0;
}